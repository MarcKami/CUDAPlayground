
#include <hip/hip_runtime.h>
/*
 * Copyright 2019-2020 Marc Martos.  All rights reserved.
 */
//
//// System includes
//#include <iostream>
//#include <stdio.h>
//#include <assert.h>
//#include <math.h>
//
//// CUDA Runtime
//#include <cuda_runtime.h>
//
//// helper functions and utilities to work with CUDA
//#include <helper_functions.h>
//#include <helper_cuda.h>
//
//// Kernel function to add the elements of two arrays
//__global__
//void add(int n, float* x, float* y) {
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int stride = blockDim.x * gridDim.x;
//	for (int i = index; i < n; i += stride)
//		y[i] = x[i] + y[i];
//}
//
//__global__
//void calcError(int n, float* y, float* err) {
//	int index = blockIdx.x * blockDim.x + threadIdx.x;
//	int stride = blockDim.x * gridDim.x;
//	for (int i = index; i < n; i += stride)
//		*err = fmax(*err, fabs(y[i] - 3.0f));
//}
//
//int main(int argc, char** argv) {
//	const int N = 1 << 20;
//	float* x, * y;
//	float s_x[N], s_y[N];
//
//	// Allocate Unified Memory � accessible from CPU or GPU
//	cudaMallocManaged(&x, N * sizeof(float));
//	cudaMallocManaged(&y, N * sizeof(float));
//
//	// initialize x and y arrays on the host
//	for (int i = 0; i < N; i++) {
//		x[i] = 1.0f;
//		y[i] = 2.0f;
//	}
//	int blockSize = 256;
//	int numBlocks = (N + blockSize - 1) / blockSize;
//	// Run kernel on 1M elements on the GPU
//	//cudaMemcpy(x, s_x, N * sizeof(float), cudaMemcpyHostToDevice);
//	add <<< numBlocks , blockSize >>> (N, x, y);
//	//cudaMemcpy(s_x, x, N * sizeof(float), cudaMemcpyDeviceToHost);
//
//	// Wait for GPU to finish before accessing on host
//	cudaDeviceSynchronize();
//
//	// Check for errors (all values should be 3.0f)
//	float* maxError;
//
//	// Allocate Unified Memory � accessible from CPU or GPU
//	cudaMallocManaged(&maxError, sizeof(float));
//
//	// Run kernel on 1M elements on the GPU
//	calcError <<< 8, 1024 >> > (N, y, maxError);
//
//	// Wait for GPU to finish before accessing on host
//	cudaDeviceSynchronize();
//	
//	std::cout << "Max error: " << *maxError << std::endl;
//
//	// Free memory
//	cudaFree(x);
//	cudaFree(y);
//	cudaFree(maxError);
//
//	return EXIT_SUCCESS;
//}